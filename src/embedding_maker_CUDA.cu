#include "hip/hip_runtime.h"
#ifndef CUDA_THINGS_H
#define CUDA_THINGS_H
#include <stdint.h>
#include <stdio.h>
#include "constants_global.h"

extern "C"{

__device__ __forceinline__ uint32_t random_uint32_t_xorshift32(uint32_t* rand_state){
    *rand_state ^= *rand_state << 13u;
    *rand_state ^= *rand_state >> 17u;
    *rand_state ^= *rand_state << 5u;
    return *rand_state;
}

// inline device funtion to calculate the squared euclidean distance between two points
__device__ __forceinline__ float cuda_euclidean_sq(float* Xi, float* Xj){
    float eucl_sq = 0.0f;
    #pragma unroll
    for (uint32_t m = 0; m < Mld; m++) {
        float diff = Xi[m] - Xj[m];
        eucl_sq += diff * diff;
    }
    return eucl_sq;
}

// inline device function that computes the simplified Cauchy kernel
// kernel function : 1. / powf(1. + eucl_sq/alpha_cauchy, alpha_cauchy);
// carefull: UNSAFE, alpha needs to be strictly > 0
__device__ __forceinline__ float cuda_cauchy_kernel(float eucl_sq, float alpha){
    return 1.0f / powf(1.0f + eucl_sq/alpha, alpha);
}




/***
 *                           _ _      _                      _            _   _             
 *                          | | |    | |                    | |          | | (_)            
 *     _ __   __ _ _ __ __ _| | | ___| |        _ __ ___  __| |_   _  ___| |_ _  ___  _ __  
 *    | '_ \ / _` | '__/ _` | | |/ _ \ |       | '__/ _ \/ _` | | | |/ __| __| |/ _ \| '_ \ 
 *    | |_) | (_| | | | (_| | | |  __/ |       | | |  __/ (_| | |_| | (__| |_| | (_) | | | |
 *    | .__/ \__,_|_|  \__,_|_|_|\___|_|       |_|  \___|\__,_|\__,_|\___|\__|_|\___/|_| |_|
 *    | |                                                                                   
 *    |_|                                                                                   
 */

__device__ void warpReduce_periodic_maxReduction_on_matrix(volatile float* matrix, uint32_t e, uint32_t prev_len, uint32_t stride, uint32_t Ncol){
    while(stride > 1u){
        prev_len = stride;
        stride   = (uint32_t) ceilf((float)prev_len * 0.5f);
        if((e + stride < prev_len)){
            #pragma unroll
            for(uint32_t m = 0u; m < Mld; m++){
                float maxval   = fmaxf(matrix[m*Ncol + stride], matrix[m*Ncol]);
                matrix[m*Ncol] = maxval; // CANNOT DO TERNARY OPERATION BECAUSE RACE CONDITION!!!!
            }
        }
    }
}

__device__ __forceinline__ void periodic_maxReduction_on_matrix(float* matrix, uint32_t Nrows, uint32_t Ncol, uint32_t period, uint32_t e){
    __syncthreads(); // just in case it wasn't done after writing
    uint32_t prev_len = 2u * period;
    uint32_t stride   = period;
    while(stride > 32u){
        prev_len = stride;
        stride   = (uint32_t) ceilf((float)prev_len * 0.5f);
        if((e + stride < prev_len)){
            #pragma unroll
            for(uint32_t m = 0u; m < Mld; m++){
                float maxval   = fmaxf(matrix[m*Ncol + stride], matrix[m*Ncol]);
                matrix[m*Ncol] = maxval; // CANNOT DO TERNARY OPERATION BECAUSE RACE CONDITION!!!!
            }
        }
        __syncthreads();
    }
    // one warp remaining: no need to sync anymore (volatile float* matrix prevents reordering)
    if(e + stride < prev_len){ 
        warpReduce_periodic_maxReduction_on_matrix(matrix, e, prev_len, stride, Ncol);}
    __syncthreads(); // this one is not necessary
}



__device__ void warpReduce_periodic_sumReduction_on_matrix(volatile float* matrix, uint32_t e, uint32_t prev_len, uint32_t stride, uint32_t Ncol){
    while(stride > 1u){
        prev_len = stride;
        stride   = (uint32_t) ceilf((float)prev_len * 0.5f);
        if((e + stride < prev_len)){
            #pragma unroll
            for(uint32_t m = 0u; m < Mld; m++){
                float to_add    = matrix[m*Ncol + stride];
                matrix[m*Ncol] += to_add; // don't use += on things on the right that can be modified by other threads
            }
        }
    }
}

// this function computes the periodic sum on a matrix of dimension (Nrows, period*Nvectors) (= (Nrows, Ncol))
// accessing the row r, and element e of the nth vector : matrix[r * Ncol + n * period + e]
/*
visual example:

| _A_  1  2  3  4  5  6  7  8  9  | _C_ 11 12 13 14 15 16 17 18 19  | 20 21 22 23 24 25 26 27 28 29 |
| _B_  2  3  4  5  6  7  8  9 10  | 11  12 13 14 15 16 17 18 19 20 | 21 22 23 24 25 26 27 28 29 30 |
| 2    3  4  5  6  7  8  9 10 11  | 12  13 14 15 16 17 18 19 20 21 | 22 23 24 25 26 27 28 29 30 31 |
| 3    4  5  6  7  8  9 10 11 12  | 13  14 15 16 17 18 19 20 21 22 | 23 24 25 26 27 28 29 30 31 32 |
| 4    5  6  7  8  9 10 11 12 13  | 14  15 16 17 18 19 20 21 22 23 | 24 25 26 27 28 29 30 31 32 33 |
| 5    6  7  8  9 10 11 12 13 14  | 15  16 17 18 19 20 21 22 23 24 | 25 26 27 28 29 30 31 32 33 34 |
| 6    7  8  9 10 11 12 13 14 15  | 16  17 18 19 20 21 22 23 24 25 | 26 27 28 29 30 31 32 33 34 35 |

--> after this function, A will contain the sum (A + 1 + 2 + ... + 9) 
                         B will contain the sum (B + 2 + 3 + ... + 10) 
                         C will contain the sum (C + 11 + 12 + ... + 19)
                        ... and so on
*/
__device__ __forceinline__ void periodic_sumReduction_on_matrix(float* matrix, uint32_t Nrows, uint32_t Ncol, uint32_t period, uint32_t e){
    __syncthreads(); // just in case it wasn't done after writing
    uint32_t prev_len = 2u * period;
    uint32_t stride   = period;
    while(stride > 32u){
        prev_len = stride;
        stride   = (uint32_t) ceilf((float)prev_len * 0.5f);
        if((e + stride < prev_len)){
            #pragma unroll
            for(uint32_t m = 0u; m < Mld; m++){
                float to_add    = matrix[m*Ncol + stride];
                matrix[m*Ncol] += to_add; // don't use += on things on the right that can be modified by other threads
            }
        }
        __syncthreads();
    }
    // one warp remaining: no need to sync anymore (volatile float* matrix prevents reordering)
    if(e + stride < prev_len){ 
        warpReduce_periodic_sumReduction_on_matrix(matrix, e, prev_len, stride, Ncol);}
    __syncthreads(); // this one is not necessary
}


/***
 *                    _                _                        _     
 *                   | |              | |                      | |    
 *      ___ _   _  __| | __ _         | | _____ _ __ _ __   ___| |___ 
 *     / __| | | |/ _` |/ _` |        | |/ / _ \ '__| '_ \ / _ \ / __|
 *    | (__| |_| | (_| | (_| |        |   <  __/ |  | | | |  __/ \__ \
 *     \___|\__,_|\__,_|\__,_|        |_|\_\___|_|  |_| |_|\___|_|___/
 *                                                                    
 *                                                                    
 */

/*
visual representation of shared memory:
        |[----- Mld -----]|    each row contains Xi
        |        .        |                                           <----   this first block is of height Ni and width Mld
        |        .        |
        |[----- Mld -----]|
        --------------------------------------
        | M | M | M | M | ... | M | M | M | M |    
        | l | l | l | l | ... | l | l | l | l |    momentum for i (m1)
        | d | d | d | d | ... | d | d | d | d |        
        --------------------------------------                        <----  this second block is of height 2Mld and width block_surface (= Ni*N_RAND)
        | M | M | M | M | ... | M | M | M | M |    
        | l | l | l | l | ... | l | l | l | l |    momentum updates for j
        | d | d | d | d | ... | d | d | d | d |    
        --------------------------------------
 */
__global__ void interactions_far(uint32_t N, float* dvc_Xld_nester, float Qdenom_EMA,\
        float alpha_cauchy, double* dvc_Qdenom_elements, float* dvc_momenta_repulsion_far,\
        uint32_t* random_numbers_size_NxRand){
    // ~~~~~~~~~~~~~~~~~~~ get i, k and j ~~~~~~~~~~~~~~~~~~~
    uint32_t Ni            = blockDim.y; // block shape: (NB_RANDOM_POINTS_FAR_REPULSION, Ni)
    uint32_t block_surface = NB_RANDOM_POINTS_FAR_REPULSION * Ni;
    uint32_t i0            = (block_surface * blockIdx.x) / NB_RANDOM_POINTS_FAR_REPULSION; // the value of the smallest i in the block
    uint32_t k             = threadIdx.x;       // block shape: (NB_RANDOM_POINTS_FAR_REPULSION, Ni)
    uint32_t i             = i0 + threadIdx.y;  // block shape: (NB_RANDOM_POINTS_FAR_REPULSION, Ni)
    if( i >= N ){return;} // out of bounds
    uint32_t tid           = threadIdx.x + threadIdx.y * NB_RANDOM_POINTS_FAR_REPULSION; // index within the block
    // get j using the random uint32_t
    uint32_t random_number = random_numbers_size_NxRand[i * NB_RANDOM_POINTS_FAR_REPULSION + k];
    uint32_t j             = random_number % N;


    // ~~~~~~~~~~~~~~~~~~~ Initialise registers: Xj ~~~~~~~~~~~~~~~~~~~

    // ~~~~~~~~~~~~~~~~~~~ Initialise shared memory ~~~~~~~~~~~~~~~~~~~

    // ~~~~~~~~~~~~~~~~~~~ now for some computations:  wij ~~~~~~~~~~~~~~~~~~~

    // ~~~~~~~~~~~~~~~~~~~ save wij for Qdenom computation (offset is already done) ~~~~~~~~~~~~~~~~~~~

    // ~~~~~~~~~~~~~~~~~~~ repulsive forces far ~~~~~~~~~~~~~~~~~~~
    // DO NOT APPLY MOMENTA ON j, ONLY i (because else we don't have a guaranteed balance on the forces)

    // ~~~~~~~~~~~~~~~~~~~ update the new seed for next iteration ~~~~~~~~~~~~~~~~~~~
    random_numbers_size_NxRand[i * NB_RANDOM_POINTS_FAR_REPULSION + k] = random_uint32_t_xorshift32(&random_numbers_size_NxRand[i * NB_RANDOM_POINTS_FAR_REPULSION + k]); // save the new random number
    // printf("old rand: %u, new rand: %u    (i %u  k %u)\n", random_number, random_numbers_size_NxRand[i * NB_RANDOM_POINTS_FAR_REPULSION + k], i, k);


    // FAIRE CTRL-F Kld : FAUT QUE RIEN N APPARAISSE DANS CE KERNEL
    // printf("ok\n");
    return;
}




/* visual representation of shared memory:
        |[----- Mld -----]|    each row contains Xi
        |        .        |                                           <----   this first block is of height Ni and width Mld
        |        .        |
        |[----- Mld -----]|
        --------------------------------------
        | M | M | M | M | ... | M | M | M | M |    
        | l | l | l | l | ... | l | l | l | l |    momentum for i (m1)
        | d | d | d | d | ... | d | d | d | d |        
        --------------------------------------                        <----  this second block is of height 2Mld and width block_surface (= Ni*Kld)
        | M | M | M | M | ... | M | M | M | M |    
        | l | l | l | l | ... | l | l | l | l |    momentum updates for j
        | d | d | d | d | ... | d | d | d | d |    
        --------------------------------------
  for this 2nd block, each thread is a column (block_surface columns in total, (ie: Ni*Kld columns))
  each row of the second block is organised as such:
        |i0,k0| i0,k1 | i0,k2 | ... | i0,k(Kld-1) | i1,k0 | i1,k1 | i1,k2 | ... | i1,k(Kld-1) | ... | i(Ni-1),k(Kld-1)|
*/
__global__ void interactions_K_LD(uint32_t N, float* dvc_Xld_nester, uint32_t* dvc_neighsLD, float Qdenom_EMA,\
        float alpha_cauchy, double* dvc_Qdenom_elements, float* dvc_momenta_repulsion, float* temporary_furthest_neighdists){
    // ~~~~~~~~~~~~~~~~~~~ get i, k and j ~~~~~~~~~~~~~~~~~~~
    uint32_t Ni            = blockDim.y; // block shape: (Kld, Ni)
    uint32_t block_surface = Kld * Ni;
    uint32_t i0            = (block_surface * blockIdx.x) / Kld; // the value of the smallest i in the block
    uint32_t k             = threadIdx.x;       // block shape: (Kld, Ni)
    uint32_t i             = i0 + threadIdx.y;  // block shape: (Kld, Ni)
    if( i >= N ){return;} // out of bounds (no guarantee that N is a multiple of Kld)
    uint32_t j             = dvc_neighsLD[i * Kld + k]; 
    uint32_t tid           = threadIdx.x + threadIdx.y * Kld; // index within the block

    // ~~~~~~~~~~~~~~~~~~~ Initialise registers: Xj ~~~~~~~~~~~~~~~~~~~
    float Xj[Mld];
    #pragma unroll
    for (uint32_t m = 0u; m < Mld; m++) { // fetch Xj from DRAM
        Xj[m] = dvc_Xld_nester[j * Mld + m];}

    // ~~~~~~~~~~~~~~~~~~~ Initialise shared memory ~~~~~~~~~~~~~~~~~~~
    extern __shared__ float smem[];
    float* Xi                   = &smem[(i - i0) * Mld];
    float* momenta_update_i_T   = &smem[Ni*Mld + tid];  // stride for changing m: block_surface
    float* momenta_update_j_T   = &smem[Ni*Mld + block_surface*Mld + tid]; // stride for changing m: block_surface
    if(k == 0){ // fetch Xi from DRAM
        #pragma unroll
        for (uint32_t m = 0u; m < Mld; m++) {
            Xi[m] = dvc_Xld_nester[i * Mld + m];}
    }
    __syncthreads();

    // ~~~~~~~~~~~~~~~~~~~ now for some computations:  wij (offset already done) ~~~~~~~~~~~~~~~~~~~
    // compute squared euclidean distance 
    float eucl_sq = cuda_euclidean_sq(Xi, Xj);
    // similarity in LD (qij = wij / Qdenom_EMA)
    float wij     = cuda_cauchy_kernel(eucl_sq, alpha_cauchy); 

    // ~~~~~~~~~~~~~~~~~~~ save wij for Qdenom computation ~~~~~~~~~~~~~~~~~~~
    dvc_Qdenom_elements[(i * Kld + k)] = (double) wij;
    
    // ~~~~~~~~~~~~~~~~~~~ repulsive forces ~~~~~~~~~~~~~~~~~~~
    // individual updates to momenta for repulsion
    // float common_repulsion_gradient_multiplier  = -(wij / Qdenom_EMA) * (2.0f * powf(wij, __frcp_rn(alpha_cauchy)));
    float common_repulsion_gradient_multiplier  = -(wij * __frcp_rn(Qdenom_EMA)) * (2.0f * powf(wij, __frcp_rn(alpha_cauchy)));

    printf("%e     %e \n", -(wij * __frcp_rn(Qdenom_EMA)), -(wij / Qdenom_EMA));

    #pragma unroll
    for(uint32_t m = 0u; m < Mld; m++){
        float gradient = (Xi[m] - Xj[m]) * common_repulsion_gradient_multiplier;
        momenta_update_i_T[m*block_surface] = -gradient; // i movement
        momenta_update_j_T[m*block_surface] =  gradient; // j movement
    }
    // aggregate the individual updates
    periodic_sumReduction_on_matrix(momenta_update_i_T, Mld, block_surface, Kld, k);
    // write to global memory for point i
    if(k == 0u){
        #pragma unroll
        for(uint32_t m = 0u; m < Mld; m++){
            atomicAdd(&dvc_momenta_repulsion[i * Mld + m], momenta_update_i_T[m*block_surface]);}
    }
    // write individual updates to j repulsion momenta
    #pragma unroll
    for(uint32_t m = 0u; m < Mld; m++){
        atomicAdd(&dvc_momenta_repulsion[j * Mld + m], momenta_update_j_T[m*block_surface]);}

    // ~~~~~~~~~~~~~~~~~~~ find the fursthest neighbour distance in LD and save it ~~~~~~~~~~~~~~~~~~~
    // start by writing eucl to shared memory
    momenta_update_i_T[0] = eucl_sq;


    __syncthreads();    remove this shiiiiiit
    float max_ = eucl_sq;    remove this shiiiiiit
    if(i == 121u && k == 0u){    remove this shiiiiiit
        for(uint32_t k2 = 0u; k2 < Kld; k2++){    remove this shiiiiiit
            printf(" %f\n", momenta_update_i_T[k2]);    remove this shiiiiiit
            if(momenta_update_i_T[k2] > max_){    remove this shiiiiiit
                max_ = momenta_update_i_T[k2];    remove this shiiiiiit
            }    remove this shiiiiiit
        }    remove this shiiiiiit
    }    remove this shiiiiiit
    __syncthreads();    remove this shiiiiiit



    // find the furthest neighbour distance in LD (parallel reduction)
    periodic_maxReduction_on_matrix(momenta_update_i_T, 1u, block_surface, Kld, k);
    // write to global memory for point i
    if(k == 0u){
        temporary_furthest_neighdists[i] = momenta_update_i_T[0];
    }




    if(i == 121u && k == 0u){    remove this shiiiiiit
        printf("  %f ==? %f\n", temporary_furthest_neighdists[i], max_);    remove this shiiiiiit
    }    remove this shiiiiiit



}


/*
grid shape : 1-d with total number of threads >= N * Khd
block shape: (Khd, Ni)

/* visual representation of shared memory:
        |[----- Mld -----]|    each row contains Xi
        |        .        |                                           <----   this first block is of height Ni and width Mld
        |        .        |
        |[----- Mld -----]|
        --------------------------------------
        | M | M | M | M | ... | M | M | M | M |    
        | l | l | l | l | ... | l | l | l | l |    momentum for i (m1)
        | d | d | d | d | ... | d | d | d | d |        
        --------------------------------------                        <----  this second block is of height 2Mld and width block_surface (= Ni*Khd)
        | M | M | M | M | ... | M | M | M | M |    
        | l | l | l | l | ... | l | l | l | l |    momentum updates for j
        | d | d | d | d | ... | d | d | d | d |    
        --------------------------------------
  for this 2nd block, each thread is a column (block_surface columns in total, (ie: Ni*Khd columns))
  each row of the second block is organised as such:
        |i0,k0| i0,k1 | i0,k2 | ... | i0,k(Khd-1) | i1,k0 | i1,k1 | i1,k2 | ... | i1,k(Khd-1) | ... | i(Ni-1),k(Khd-1)|
*/
__global__ void interactions_K_HD(uint32_t N, float* dvc_Pij, float* dvc_Xld_nester,\
        uint32_t* dvc_neighsHD, float* furthest_neighdists_LD, float Qdenom_EMA,\
        float alpha_cauchy, double* dvc_Qdenom_elements, float* dvc_momenta_attraction,\
        float* dvc_momenta_repulsion){
    // ~~~~~~~~~~~~~~~~~~~ get i, k and j ~~~~~~~~~~~~~~~~~~~
    uint32_t Khd           = blockDim.x; // block shape: (Khd, Ni);  Khd is guaranteed to be >= 32u
    uint32_t Ni            = blockDim.y; // block shape: (Khd, Ni)
    uint32_t block_surface = blockDim.x * blockDim.y;
    uint32_t i0            = (block_surface * blockIdx.x) / Khd; // the value of the smallest i in the block
    uint32_t k             = threadIdx.x;
    uint32_t i             = i0 + threadIdx.y;
    if( i >= N ){return;} // out of bounds (no guarantee that N is a multiple of Ni)
    uint32_t j             = dvc_neighsHD[i * Khd + k]; 
    uint32_t tid           = threadIdx.x + threadIdx.y * Khd; // index within the block

    // ~~~~~~~~~~~~~~~~~~~ Initialise registers: Xj and furthest LDneighdists for i and j ~~~~~~~~~~~~~~~~~~~
    float furthest_LDneighdist_j = __ldg(&furthest_neighdists_LD[j]);
    float Xj[Mld];
    #pragma unroll
    for (uint32_t m = 0u; m < Mld; m++) { // fetch Xj from DRAM
        Xj[m] = dvc_Xld_nester[j * Mld + m];}
    float furthest_LDneighdist_i = __ldg(&furthest_neighdists_LD[i]);

    // ~~~~~~~~~~~~~~~~~~~ Initialise shared memory ~~~~~~~~~~~~~~~~~~~
    extern __shared__ float smem[];
    float* Xi                   = &smem[(i - i0) * Mld];
    float* momenta_update_i_T   = &smem[Ni*Mld + tid];  // stride for changing m: block_surface
    float* momenta_update_j_T   = &smem[Ni*Mld + block_surface*Mld + tid]; // stride for changing m: block_surface
    if(k == 0){ // fetch Xi from DRAM
        #pragma unroll
        for (uint32_t m = 0u; m < Mld; m++) {
            Xi[m] = dvc_Xld_nester[i * Mld + m];}
    }
    __syncthreads();

    // ~~~~~~~~~~~~~~~~~~~ now for some computations: prepare pij and wij ~~~~~~~~~~~~~~~~~~~
    // compute squared euclidean distance 
    float eucl_sq = cuda_euclidean_sq(Xi, Xj);
    // similarity in HD
    float pij     = __ldg(&dvc_Pij[i * Khd + k]);
    // similarity in LD (qij = wij / Qdenom_EMA)
    float wij     = cuda_cauchy_kernel(eucl_sq, alpha_cauchy); 

    // ~~~~~~~~~~~~~~~~~~~ save wij for Qdenom computation ~~~~~~~~~~~~~~~~~~~
    dvc_Qdenom_elements[(i * Khd + k)] = (double) wij; 

    // ~~~~~~~~~~~~~~~~~~~ attractive forces ~~~~~~~~~~~~~~~~~~~
    // individual updates to momenta for attraction
    float powerthing = 2.0f * powf(wij, __frcp_rn(alpha_cauchy));
    float common_attraction_gradient_multiplier =  pij * powerthing;
    #pragma unroll
    for(uint32_t m = 0u; m < Mld; m++){
        float gradient = (Xi[m] - Xj[m]) * common_attraction_gradient_multiplier;
        momenta_update_i_T[m*block_surface] = -gradient; // i movement
        momenta_update_j_T[m*block_surface] =  gradient; // j movement
    }
    // aggregate the individual updates
    periodic_sumReduction_on_matrix(momenta_update_i_T, Mld, block_surface, Khd, k);
    // write aggregated to global memory for point i
    if(k == 0u){
        #pragma unroll
        for(uint32_t m = 0u; m < Mld; m++){
            atomicAdd(&dvc_momenta_attraction[i * Mld + m], momenta_update_i_T[m*block_surface]);}
    }
    // write individual updates to j attraction momenta
    #pragma unroll
    for(uint32_t m = 0u; m < Mld; m++){
        atomicAdd(&dvc_momenta_attraction[j * Mld + m], momenta_update_j_T[m*block_surface]);}

    // ~~~~~~~~~~~~~~~~~~~ repulsive forces ~~~~~~~~~~~~~~~~~~~
    // individual updates to momenta for repulsion
    bool do_repulsion = eucl_sq > furthest_LDneighdist_i && eucl_sq > furthest_LDneighdist_j; // do repulsion if not LD neighbours. 
    if(do_repulsion){ // the  conditional is annoying because there is no structure in the decision to do repulsion or not: x2 time taken
        // float common_repulsion_gradient_multiplier  = -(wij / Qdenom_EMA) * powerthing;
        float common_repulsion_gradient_multiplier  = -(wij * __frcp_rn(Qdenom_EMA)) * powerthing;
        #pragma unroll
        for(uint32_t m = 0u; m < Mld; m++){
            float gradient = (Xi[m] - Xj[m]) * common_repulsion_gradient_multiplier;
            momenta_update_i_T[m*block_surface] = -gradient; // i movement
            momenta_update_j_T[m*block_surface] =  gradient; // j movement
        }
    }
    else{ // important to set to zero, else the aggregation will be wrong
        #pragma unroll
        for(uint32_t m = 0u; m < Mld; m++){
            momenta_update_i_T[m*block_surface] = 0.0f;
            momenta_update_j_T[m*block_surface] = 0.0f;
        }
    }
    // aggregate the individual updates
    periodic_sumReduction_on_matrix(momenta_update_i_T, Mld, block_surface, Khd, k);
    // write to global memory for point i
    if(k == 0u){
        #pragma unroll
        for(uint32_t m = 0u; m < Mld; m++){
            atomicAdd(&dvc_momenta_repulsion[i * Mld + m], momenta_update_i_T[m*block_surface]);}
    }
    // write individual updates to j repulsion momenta
    #pragma unroll
    for(uint32_t m = 0u; m < Mld; m++){
        atomicAdd(&dvc_momenta_repulsion[j * Mld + m], momenta_update_j_T[m*block_surface]);}
}




















void fill_raw_momenta_launch_cuda(hipStream_t stream_HD, hipStream_t stream_LD, hipStream_t stream_FAR,\
     uint32_t* Kern_HD_blockshape, uint32_t* Kern_HD_gridshape,uint32_t* Kern_LD_blockshape, uint32_t* Kern_LD_gridshape,uint32_t* Kern_FAR_blockshape, uint32_t* Kern_FAR_gridshape,\
      uint32_t N, uint32_t Khd, float* dvc_Pij,\
      float* dvc_Xld_nester, uint32_t* dvc_neighsHD, uint32_t* dvc_neighsLD, float* furthest_neighdists_LD, float Qdenom_EMA,\
       float alpha_cauchy, double* dvc_Qdenom_elements,\
        float* dvc_momenta_attraction, float* dvc_momenta_repulsion, float* dvc_momenta_repulsion_far, float* temporary_furthest_neighdists,\
         uint32_t* random_numbers_size_NxRand){
    
    // ~~~~~~~~~  clear momenta (async)  ~~~~~~~~~
    hipMemsetAsync(dvc_momenta_attraction, 0, N * Mld * sizeof(float), stream_HD);
    hipMemsetAsync(dvc_momenta_repulsion, 0, N * Mld * sizeof(float), stream_LD);
    hipMemsetAsync(dvc_momenta_repulsion_far, 0, N * Mld * sizeof(float), stream_FAR);

    // ~~~~~~~~~  prepare kernel calls ~~~~~~~~~
    // Kernel 1: HD neighbours
    uint32_t Kern_HD_block_surface = Kern_HD_blockshape[0] * Kern_HD_blockshape[1]; // N threads per block
    // if((Kern_HD_block_surface % 32) != 0){printf("\n\nError: block size should be a multiple of 32\n");return;}
    uint32_t Kern_HD_sharedMemorySize = (uint32_t) (sizeof(float) * ((Kern_HD_blockshape[1] * Mld) + (Kern_HD_block_surface * (2u * Mld))));
    dim3 Kern_HD_grid(Kern_HD_gridshape[0], Kern_HD_gridshape[1]);
    dim3 Kern_HD_block(Kern_HD_blockshape[0], Kern_HD_blockshape[1]);
    // Kernel 2: LD neighbours
    uint32_t Kern_LD_block_surface = Kern_LD_blockshape[0] * Kern_LD_blockshape[1]; // N threads per block
    uint32_t Kern_LD_sharedMemorySize = (uint32_t) (sizeof(float) * ((Kern_LD_blockshape[1] * Mld) + (Kern_LD_block_surface * (2u * Mld))));
    dim3 Kern_LD_grid(Kern_LD_gridshape[0], Kern_LD_gridshape[1]);
    dim3 Kern_LD_block(Kern_LD_blockshape[0], Kern_LD_blockshape[1]);
    // Kernel 3: FAR neighbours
    uint32_t Kern_FAR_block_surface = Kern_FAR_blockshape[0] * Kern_FAR_blockshape[1]; // N threads per block
    uint32_t Kern_FAR_sharedMemorySize = (uint32_t) (sizeof(float) * ((Kern_FAR_blockshape[1] * Mld) + (Kern_FAR_block_surface * (2u * Mld))));
    dim3 Kern_FAR_grid(Kern_FAR_gridshape[0], Kern_FAR_gridshape[1]);
    dim3 Kern_FAR_block(Kern_FAR_blockshape[0], Kern_FAR_blockshape[1]);

    
    // ~~~~~~~~~  launch kernels (and wait for async memset to finish)  ~~~~~~~~~
    // kernel 1 : HD neighbours
    hipStreamSynchronize(stream_HD); // wait for the momenta to clear
    interactions_K_HD<<<Kern_HD_grid, Kern_HD_block, Kern_HD_sharedMemorySize, stream_HD>>>(N, dvc_Pij, dvc_Xld_nester, dvc_neighsHD, furthest_neighdists_LD, Qdenom_EMA, alpha_cauchy, dvc_Qdenom_elements, dvc_momenta_attraction, dvc_momenta_repulsion);// launch the kernel 1
    hipError_t err1 = hipGetLastError();
    if (err1 != hipSuccess) {printf("Error in kernel 1: %s\n", hipGetErrorString(err1));}
    // kernel 2 : LD neighbours
    hipStreamSynchronize(stream_LD); // wait for the momenta to clear
    interactions_K_LD<<<Kern_LD_grid, Kern_LD_block, Kern_LD_sharedMemorySize, stream_LD>>>(N, dvc_Xld_nester, dvc_neighsLD, Qdenom_EMA, alpha_cauchy, &dvc_Qdenom_elements[N*Khd], dvc_momenta_repulsion, temporary_furthest_neighdists);// launch the kernel 2
    hipError_t err2 = hipGetLastError();
    if (err2 != hipSuccess) {printf("Error in kernel 2: %s\n", hipGetErrorString(err2));}
    // kernel 3 : FAR neighbours
    hipStreamSynchronize(stream_FAR); // wait for the momenta to clear
    interactions_far<<<Kern_FAR_grid, Kern_FAR_block, Kern_FAR_sharedMemorySize, stream_FAR>>>(N, dvc_Xld_nester, Qdenom_EMA, alpha_cauchy, &dvc_Qdenom_elements[N*Khd + N*Kld], dvc_momenta_repulsion_far, random_numbers_size_NxRand);// launch the kernel 3
    hipError_t err3 = hipGetLastError();
    if (err3 != hipSuccess) {printf("Error in kernel 3: %s\n", hipGetErrorString(err3));}

    

    // ~~~~~~~~~~~  memcpy for furthest_neighdists_LD  ~~~~~~~~~
    // wait for the 1st kernel to finish (because it uses furthest_neighdists_LD)
    hipStreamSynchronize(stream_HD);
    // wait for the 2nd kernel to finish (because it writes to temporary_furthest_neighdists)
    hipStreamSynchronize(stream_LD);
    // do a memcpy from temporary_furthest_neighdists to furthest_neighdists_LD 
    hipMemcpyAsync(furthest_neighdists_LD, temporary_furthest_neighdists, N * sizeof(float), hipMemcpyDeviceToDevice, stream_LD);
    

    // ~~~~~~~~~~~  sync streams  ~~~~~~~~~
    hipStreamSynchronize(stream_FAR);
    hipStreamSynchronize(stream_LD);


   
    // TODO: ascend to godhood by using pretch CUDA instruction in assembly (Fermi architecture)
    // the prefetch instruction is used to load the data from global memory to the L2 cache
}

}

#endif